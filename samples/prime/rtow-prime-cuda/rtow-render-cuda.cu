#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2025 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "World.h"
#include "FrameBuffer.h"

#include <random>
#include <cmath>
#include <random>
#include <iostream>
#include <iomanip>
#include <chrono>
#include <memory>
#include <thread>

namespace samples {
  namespace device {
  
    struct PathState {
      vec3f  weight;
      Random random;
    };

    struct LaunchState {
      // one entry per pixel:
      Ray          *rays;
      Hit          *hits;
      device::PathState        *paths;

      // list of which rays are active
      int                      *activeRaysIn;
      int                      *activeRaysOut;
      // device-size *counter* of how many rays are active (serves as
      // atomic to write into d_activeRays)
      int                      *pNumActive;
    };
  
    inline __device__
    vec3f missColor(const vec3f &rayDir)
    {
      const float t = 0.5f*(rayDir.y + 1.0f);
      const vec3f c = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
      return c;
    }

    __global__
    void waveFrontPathCreate(Scene       world,
                             LaunchState launch,
                             int         sampleID)
    {
      const vec2i pixelID
        = vec2i(threadIdx)
        + vec2i(blockIdx)
        * vec2i(blockDim);
    
      if (pixelID.x >= world.fbSize.x) return;
      if (pixelID.y >= world.fbSize.y) return;
    
      const int tID = pixelID.x+world.fbSize.x*pixelID.y;
      
      PathState &path = launch.paths[tID];
      Ray       &ray  = launch.rays[tID];
      Hit       &hit  = launch.hits[tID];
        
      path.random.init(pixelID.x+sampleID*world.fbSize.x,
                       pixelID.y+sampleID*world.fbSize.y);
      path.weight = vec3f(1.f);
      
      const float u = float(pixelID.x + path.random()) / float(world.fbSize.x);
      const float v = float(pixelID.y + path.random()) / float(world.fbSize.y);
      ray = world.camera.generateRay(u,v,path.random);
      hit.primID = -1;
      hit.instID = -1;
    
      // and finally: initially every ray is active:
      launch.activeRaysOut[tID] = tID;
    }
    
    __global__
    void waveFrontAccumulate(Scene world,
                             LaunchState launch)
    {
      const vec2i pixelID
        = vec2i(threadIdx)
        + vec2i(blockIdx)
        * vec2i(blockDim);
    
      if (pixelID.x >= world.fbSize.x) return;
      if (pixelID.y >= world.fbSize.y) return;
    
      const int tID = pixelID.x+world.fbSize.x*pixelID.y;
      
      PathState &path        = launch.paths[tID];
        
      const vec3f addtl = (1.f/world.numSamplesPerPixel)*path.weight;
      vec3f &fb = world.fb[tID];
      atomicAdd(&fb.x,addtl.x);
      atomicAdd(&fb.y,addtl.y);
      atomicAdd(&fb.z,addtl.z);
    }
    

    __global__
    void waveFrontShade(int    numActive,
                        Scene  world,
                        LaunchState launch,
                        int    depth,
                        int    maxDepth)
    {
      const vec2i pixelID
        = vec2i(threadIdx)
        + vec2i(blockIdx)
        * vec2i(blockDim);
      const int linearThreadID = pixelID.x+world.fbSize.x*pixelID.y;
      if (linearThreadID >= numActive)
        return;
    
      const int rayID = launch.activeRaysIn[linearThreadID];
    
      PathState       &path   = launch.paths[rayID];
      Ray &ray    = launch.rays[rayID];
      Hit &hit    = launch.hits[rayID];

      if (hit.primID < 0) {
        if (depth == 0) 
          path.weight = missColor(normalize(ray.direction));
        return;
      }
    
      if (depth >= maxDepth) {
        path.weight = 0.f;
        return;
      }
      int instID = hit.instID;
      affine3f xfm = world.transforms[instID];
      auto &random = path.random;

      ScatterEvent scatter;
      vec3f &org = (vec3f&)ray.origin;
      vec3f &dir = (vec3f&)ray.direction;
      scatter.inDir = normalize(dir);
      int materialID = 0;
      // geomID==0 : triangles
      Triangle tri = world.triangles[hit.primID];
      tri.A = xfmPoint(xfm,tri.A);
      tri.B = xfmPoint(xfm,tri.B);
      tri.C = xfmPoint(xfm,tri.C);
      const float u = hit.u;
      const float v = hit.v;
#if HAVE_SHADING_NORMALS
      scatter.N
        = normalize((1.f-u-v) * tri.Na
                    + u       * tri.Nb
                    + v       * tri.Nc);
      if (dot(scatter.N,ray.direction) >= 0.f)
        scatter.N = normalize(cross(tri.B-tri.A,tri.C-tri.A));
#else
      scatter.N = normalize(cross(tri.B-tri.A,tri.C-tri.A));
#endif
      scatter.P =
        (1.f-u-v) * tri.A
        + u       * tri.B
        + v       * tri.C;
      materialID = world.materialIDs[instID];
      auto &material = world.materials[materialID];
      if (!material.scatter(scatter,random))
        // path lost .... ugh.
        return;
      
      org = scatter.out_org;
      dir = scatter.out_dir;
      ray.tmin = 1e-4f;
      ray.tmax = 1e20f;
      path.weight *= scatter.attenuation;
      int activeSlot = atomicAdd(launch.pNumActive,1);
      launch.activeRaysOut[activeSlot] = rayID;
    }
    
  } // ::device
  




  struct WaveFront {
    WaveFront(std::shared_ptr<World> world,
              const vec2i fbSize)
    {
      numAllocated = fbSize.x*fbSize.y;
      hipMalloc(&d_launch.rays,numAllocated*sizeof(Ray));
      hipMalloc(&d_launch.hits,numAllocated*sizeof(Hit));
      hipMalloc(&d_launch.paths,numAllocated*sizeof(device::PathState));
      hipMalloc(&d_launch.activeRaysIn,numAllocated*sizeof(int));
      hipMalloc(&d_launch.activeRaysOut,numAllocated*sizeof(int));
      hipMalloc(&d_launch.pNumActive,sizeof(int));
    }

    device::LaunchState d_launch;
    int numAllocated = 0;
  };

  void renderOneSample(const World &world,
                       const device::Scene &devScene,
                       WaveFront &waveFront,
                       // owl::prime::TraceContext *prime,
                       int sampleID, int numSamplesPerPixel)
  {
    hipStream_t     stream = 0;//prime->getStream();
    vec2i tileSize = 16;

    device::LaunchState &launch = waveFront.d_launch;
    {
      vec2i numTiles = divRoundUp(devScene.fbSize,tileSize);
      device::waveFrontPathCreate<<<numTiles,tileSize,0,stream>>>
        (devScene,launch,sampleID);
    }

    const int numActiveAtStart = waveFront.numAllocated;
    int numActive = numActiveAtStart;
    for (int depth=0;numActive > 0;depth++) {
      
      // ------------------------------------------------------------------
      // trace active rays
      // ------------------------------------------------------------------
      std::swap(launch.activeRaysIn,launch.activeRaysOut);
      // owl::prime::RequestHandle query
      //   = prime->findClosestHit(launch.rays,
      //                           launch.hits,
      //                           launch.activeRaysIn,
      //                           numActive);
      opTraceIndexed(world.model,
                     numActive,
                     (int32_t*)launch.activeRaysIn,
                     numActiveAtStart,
                     (OPRay *)launch.rays,
                     (OPHit *)launch.hits,
                     0);
              
      
      // ------------------------------------------------------------------
      // shade active rays
      // ------------------------------------------------------------------
      // re-set numactive counter
      int zero = 0;
      hipMemcpyAsync(launch.pNumActive,&zero,sizeof(zero),
                      hipMemcpyHostToDevice,stream);
      vec2i numTiles = divRoundUp(devScene.fbSize,tileSize);
      device::waveFrontShade<<<numTiles,tileSize,0,stream>>>
        (numActive,devScene,launch,depth,devScene.maxPathLength); 
      // and read new active counter
      hipMemcpyAsync(&numActive,launch.pNumActive,sizeof(zero),
                      hipMemcpyDeviceToHost,stream);
      hipStreamSynchronize(stream);
    }
    
    // ------------------------------------------------------------------
    // and finally, accumulate all:
    // ------------------------------------------------------------------
    {
      vec2i numTiles = divRoundUp(devScene.fbSize,tileSize);
      device::waveFrontAccumulate<<<numTiles,tileSize,0,stream>>>
        (devScene,launch);
    }
  }
    
  void renderFrame(std::shared_ptr<World> world,
                   std::shared_ptr<Camera> camera,
                   std::shared_ptr<FrameBuffer> fb,
                   int numSamplesPerPixel,
                   int maxPathLength// ,
                   // int numThreads
                   )
  {
    // ------------------------------------------------------------------
    // first, upload scene
    // ------------------------------------------------------------------

    device::Scene scene;
    scene.numSamplesPerPixel = numSamplesPerPixel;
    scene.maxPathLength = maxPathLength;

    // upload materials
    hipMalloc(&scene.materials,world->materials.size()*sizeof(world->materials[0]));
    hipMemcpy(scene.materials,world->materials.data(),
               world->materials.size()*sizeof(world->materials[0]),
               hipMemcpyHostToDevice);

    // upload triangles
    hipMalloc(&scene.triangles,world->triangles.size()*sizeof(world->triangles[0]));
    hipMemcpy(scene.triangles,world->triangles.data(),
               world->triangles.size()*sizeof(world->triangles[0]),
               hipMemcpyHostToDevice);

    // upload materialIDs
    hipMalloc(&scene.materialIDs,world->materialIDs.size()*sizeof(world->materialIDs[0]));
    hipMemcpy(scene.materialIDs,world->materialIDs.data(),
               world->materialIDs.size()*sizeof(world->materialIDs[0]),
               hipMemcpyHostToDevice);

    // upload transforms
    hipMalloc(&scene.transforms,world->transforms.size()*sizeof(world->transforms[0]));
    hipMemcpy(scene.transforms,world->transforms.data(),
               world->transforms.size()*sizeof(world->transforms[0]),
               hipMemcpyHostToDevice);
    
    scene.camera = *camera;
      
    // ------------------------------------------------------------------
    // create (and clear) device-side frame buffer
    // ------------------------------------------------------------------
    scene.fbSize = fb->size;
    hipMalloc(&scene.fb,fb->size.x*fb->size.y*sizeof(vec3f));
    hipMemset(scene.fb,0,fb->size.x*fb->size.y*sizeof(vec3f));

    WaveFront waveFront(world,fb->size);
    const vec2i fbSize = fb->size;
    for (int sampleID=0;
         sampleID<numSamplesPerPixel;
         sampleID+=1) 
      renderOneSample(*world,scene,waveFront,
                      sampleID,numSamplesPerPixel);
    
    hipDeviceSynchronize();
    hipMemcpy(fb->pixels.data(),scene.fb,
               fb->size.x*fb->size.y*sizeof(vec3f),
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }

  void renderFrame(std::shared_ptr<World> world,
                   std::shared_ptr<Camera> camera,
                   std::shared_ptr<FrameBuffer> fb,
                   int numSamplesPerPixel,
                   int maxPathLength);
  
  int rtow_CUDA(int ac, const char **av)
  {
    int Nx = 800, Ny = 600;
    int spp = 128;
    int maxPathLength = 3;
    
    for (int i=1;i<ac;i++) {
      const std::string arg = av[i];
      if (arg == "-fast" || arg == "-lq") {
        Nx = 800;
        Ny = 600;
        spp  = 16;
      } else if (arg == "--final" || arg == "-hq") {
        Nx = 2*800;
        Ny = 2*600;
        spp  = 1024;
      } else if (arg == "--size") {
        Nx = std::stoi(av[++i]);
        Ny = std::stoi(av[++i]);
      } else if (arg == "--rec" || arg == "--bounces" || arg == "--max-path-length") {
        maxPathLength = std::stoi(av[++i]);;
      } else if (arg == "--spp" || arg == "-spp") {
        spp = std::stoi(av[++i]);;
      } else throw std::runtime_error("unknown arg " +arg);
    }
    
    // create - and set - the camera
    const vec3f lookfrom(13.f, 2.f, 3.f);
    const vec3f lookat(0.f, 0.f, 0.f);
    std::shared_ptr<Camera>
      camera = std::make_shared<Camera>(lookfrom,
                                        lookat,
                                        /* up */ vec3f(0.f, 1.f, 0.f),
                                        /* fovy, in degrees */ 20.f,
                                        /* aspect */ float(Nx) / float(Ny),
                                        /* aperture */ 0.1f,
                                        /* dist to focus: */ 10.f);
    
    // create a frame buffer
    std::shared_ptr<FrameBuffer> fb
      = std::make_shared<FrameBuffer>(vec2i(Nx, Ny));
    Random random;
    std::shared_ptr<World> world
      = createScene(random);
    world->finalize();
    
    // render the frame (and time it)
    auto t0 = std::chrono::system_clock::now();
    renderFrame(world,camera,fb,
                spp,maxPathLength// ,
                // numThreads
                );
    auto t1 = std::chrono::system_clock::now();
    std::cout << "done rendering, which took "
              << std::setprecision(4) << std::chrono::duration<double>(t1-t0).count()
              << " seconds (for " << spp
              << " paths per pixel)" << std::endl;
       
    savePNG("finalChapter.png",*fb);

    // ... done.
    return 0;
  }

}
  

int main(int ac, const char **av)
{ return samples::rtow_CUDA(ac,av); }
