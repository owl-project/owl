#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2021 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "InstanceGroup.h"
#include "Context.h"
#include "CUDADriver.h"

#define LOG(message)                                    \
  if (Context::logging())                               \
    std::cout << "#owl(" << device->ID << "): "         \
              << message                                \
              << std::endl

#define LOG_OK(message)                                         \
  if (Context::logging())                                       \
    std::cout << OWL_TERMINAL_GREEN                             \
              << "#owl(" << device->ID << "): "                 \
              << message << OWL_TERMINAL_DEFAULT << std::endl


namespace owl {

  /*! constructor */
  InstanceGroup::DeviceData::DeviceData(const DeviceContext::SP &device)
    : Group::DeviceData(device)
  {};

  InstanceGroup::InstanceGroup(Context *const context,
                               size_t _numChildren,
                               Group::SP *groups,
                               unsigned int _buildFlags,
                               bool _useInstanceProgram)
    : Group(context,context->groups),
      numChildren(_numChildren),
      buildFlags( (_buildFlags > 0) ? _buildFlags : defaultBuildFlags),
      useInstanceProgram(_useInstanceProgram)
  {
    if (!useInstanceProgram) {
      children.resize(numChildren);
      std::vector<uint32_t> childIDs;
      if (groups) {
        childIDs.resize(numChildren);
        for (size_t i=0;i<numChildren;i++) {
          assert(groups[i]);
          children[i] = groups[i];
          childIDs[i] = groups[i]->ID;
        }
      }

      // TODO: refactor below... doesn't really work with instance programs, since 
      // transforms will be 0 in size then, regardless of if motion blur is enabled or not
      transforms[0].resize(children.size());
      // do NOT automatically resize transforms[0] - need these only if
      // we use motion blur for this object
    }
  }
  
  
  /*! pretty-printer, for printf-debugging */
  std::string InstanceGroup::toString() const
  {
    return "InstanceGroup";
  }
  
  /*! creates the device-specific data for this group */
  RegisteredObject::DeviceData::SP InstanceGroup::createOn(const DeviceContext::SP &device) 
  {
    auto dd = std::make_shared<DeviceData>(device);
    return dd;
  }
  
  /*! set transformation matrix of given child */
  void InstanceGroup::setTransform(size_t childID,
                                   const affine3f &xfm)
  {
    if (useInstanceProgram)
      OWL_RAISE("setting children on instance group with instance program must be done on device");
    assert(childID < children.size());
    transforms[0][childID] = xfm;
  }

  void InstanceGroup::setTransforms(uint32_t timeStep,
                                    const float *floatsForThisStimeStep,
                                    OWLMatrixFormat matrixFormat)
  {
    if (useInstanceProgram) 
      OWL_RAISE("setting transforms on instance group with instance program must be done on device");
    switch(matrixFormat) {
    case OWL_MATRIX_FORMAT_OWL: {
      transforms[timeStep].resize(children.size());
      memcpy((char*)transforms[timeStep].data(),floatsForThisStimeStep,
             children.size()*sizeof(affine3f));
    } break;
    default:
      OWL_RAISE("used matrix format not yet implemented for"
                " InstanceGroup::setTransforms");
    };
  }

  /* set instance IDs to use for the children - MUST be an array of children.size() items */
  void InstanceGroup::setInstanceIDs(const uint32_t *_instanceIDs)
  {
    if (useInstanceProgram)
      OWL_RAISE("setting instance IDs on instance group with instance program must be done on device");
    instanceIDs.resize(children.size());
    std::copy(_instanceIDs,_instanceIDs+instanceIDs.size(),instanceIDs.data());
  }

  /* set visibility masks to use for the children - MUST be an array of children.size() items */
  void InstanceGroup::setVisibilityMasks(const uint8_t *_visibilityMasks)
  {
    if (useInstanceProgram)
      OWL_RAISE("setting visibility masks on instance group with instance program must be done on device");
    visibilityMasks.resize(children.size());
    std::copy(_visibilityMasks,_visibilityMasks+visibilityMasks.size(),visibilityMasks.data());
  }

  void InstanceGroup::setChild(size_t childID, Group::SP child)
  {
    if (useInstanceProgram)
      OWL_RAISE("setting children on instance group with instance program must be done on device");

    assert(childID < numChildren);
    children[childID] = child;
  }

  void InstanceGroup::setInstanceProg(Module::SP module,
                                      const std::string &progName)
  {
    if (!useInstanceProgram) 
      OWL_RAISE("trying to set instance program on instance group that was not created" 
        "with useInstanceProgram=true");
    this->instanceProg.progName = progName;
    this->instanceProg.module   = module;
  }

  void InstanceGroup::setMotionInstanceProg(Module::SP module,
                                            const std::string &progName)
  {
    if (!useInstanceProgram) 
      OWL_RAISE("trying to set instance program on instance group that was not created" 
        "with useInstanceProgram=true");
    this->motionInstanceProg.progName = progName;
    this->motionInstanceProg.module   = module;
  }

  /*! build the CUDA instance program kernel (if instance prog is set) */
  void InstanceGroup::buildInstanceProg()
  {
    if (!instanceProg.module) return;
    
    Module::SP module = instanceProg.module;
    assert(module);

    for (auto device : context->getDevices()) {
      LOG("building instance function ....");
      SetActiveGPU forLifeTime(device);
      auto &typeDD = getDD(device);
      auto &moduleDD = module->getDD(device);
      
      assert(moduleDD.computeModule);

      const std::string annotatedProgName
        = std::string("__instanceFuncKernel__")
        + instanceProg.progName;
    
      hipError_t rc = _cuModuleGetFunction(&typeDD.instanceFuncKernel,
                                        moduleDD.computeModule,
                                        annotatedProgName.c_str());
      
      switch(rc) {
      case hipSuccess:
        /* all OK, nothing to do */
        LOG_OK("found instance function " << annotatedProgName << " ... perfect!");
        break;
      case hipErrorNotFound:
        OWL_RAISE("in "+std::string(__PRETTY_FUNCTION__)
                  +": could not find OPTIX_INSTANCE_PROGRAM("
                  +instanceProg.progName+")");
      default:
        const char *errName = 0;
        _cuGetErrorName(rc,&errName);
        OWL_RAISE("unknown CUDA error when building instance program kernel"
                  +std::string(errName));
      }
    }
  }

  /*! build the CUDA motion instance program kernel (if motion instance prog is set) */
  void InstanceGroup::buildMotionInstanceProg()
  {
    if (!motionInstanceProg.module) return;
    
    Module::SP module = motionInstanceProg.module;
    assert(module);

    for (auto device : context->getDevices()) {
      LOG("building motion instance function ....");
      SetActiveGPU forLifeTime(device);
      auto &typeDD = getDD(device);
      auto &moduleDD = module->getDD(device);
      
      assert(moduleDD.computeModule);

      const std::string annotatedProgName
        = std::string("__motionInstanceFuncKernel__")
        + motionInstanceProg.progName;
    
      hipError_t rc = _cuModuleGetFunction(&typeDD.motionInstanceFuncKernel,
                                        moduleDD.computeModule,
                                        annotatedProgName.c_str());
      
      switch(rc) {
      case hipSuccess:
        /* all OK, nothing to do */
        LOG_OK("found motion instance function " << annotatedProgName << " ... perfect!");
        break;
      case hipErrorNotFound:
        OWL_RAISE("in "+std::string(__PRETTY_FUNCTION__)
                  +": could not find OPTIX_MOTION_INSTANCE_PROGRAM("
                  +motionInstanceProg.progName+")");
      default:
        const char *errName = 0;
        _cuGetErrorName(rc,&errName);
        OWL_RAISE("unknown CUDA error when building motion instance program kernel"
                  +std::string(errName));
      }
    }
  }

  void InstanceGroup::buildAccel(LaunchParams::SP launchParams)
  {
    for (auto device : context->getDevices())
      if (!useInstanceProgram && transforms[1].empty())
        staticBuildOn<true>(device);
      else if (!useInstanceProgram)
        motionBlurBuildOn<true>(device);
      else if (useInstanceProgram && transforms[1].empty())
        staticDeviceBuildOn<true>(device, launchParams);
      else if (useInstanceProgram) {
        motionBlurDeviceBuildOn<true>(device, launchParams);
      }
      else 
        OWL_RAISE("unknown instance group build type");
  }
  
  void InstanceGroup::refitAccel(LaunchParams::SP launchParams)
  {
    for (auto device : context->getDevices())
      if (!useInstanceProgram && transforms[1].empty())
        staticBuildOn<false>(device);
      else if (!useInstanceProgram)
        motionBlurBuildOn<false>(device);
      else if (useInstanceProgram && transforms[1].empty())
        staticDeviceBuildOn<false>(device, launchParams);
      else if (useInstanceProgram)
        motionBlurDeviceBuildOn<false>(device, launchParams);
  }

  template<bool FULL_REBUILD>
  void InstanceGroup::staticBuildOn(const DeviceContext::SP &device) 
  {
    DeviceData &dd = getDD(device);
    auto optixContext = device->optixContext;

    SetActiveGPU forLifeTime(device);
    LOG("building instance accel over "
        << children.size() << " groups");

    // ==================================================================
    // sanity check that that many instances are actualy allowed by optix:
    // ==================================================================
    uint32_t maxInstsPerIAS = 0;
    optixDeviceContextGetProperty
      (optixContext,
       OPTIX_DEVICE_PROPERTY_LIMIT_MAX_INSTANCES_PER_IAS,
       &maxInstsPerIAS,
       sizeof(maxInstsPerIAS));
      
    if (children.size() > maxInstsPerIAS)
      throw std::runtime_error("number of children in instance group exceeds "
                               "OptiX's MAX_INSTANCES_PER_IAS limit");

    if (!FULL_REBUILD && !(buildFlags & OPTIX_BUILD_FLAG_ALLOW_UPDATE))
      throw std::runtime_error("trying to refit an accel struct that was not built with OPTIX_BUILD_FLAG_ALLOW_UPDATE");
    
    if (FULL_REBUILD) {
      dd.memFinal = 0;
      dd.memPeak = 0;
    }
   

    // ==================================================================
    // create instance build inputs
    // ==================================================================
    OptixBuildInput              instanceInput  {};
    OptixAccelBuildOptions       accelOptions   {};
    
    //! the N build inputs that go into the builder
    std::vector<OptixInstance>   optixInstances(children.size());

    // now go over all children to set up the buildinputs
    for (size_t childID=0;childID<children.size();childID++) {
      Group::SP child = children[childID];
      assert(child);

      assert(transforms[1].empty());
      const affine3f xfm = transforms[0][childID];

      OptixInstance oi = {};
      oi.transform[0*4+0]  = xfm.l.vx.x;
      oi.transform[0*4+1]  = xfm.l.vy.x;
      oi.transform[0*4+2]  = xfm.l.vz.x;
      oi.transform[0*4+3]  = xfm.p.x;
        
      oi.transform[1*4+0]  = xfm.l.vx.y;
      oi.transform[1*4+1]  = xfm.l.vy.y;
      oi.transform[1*4+2]  = xfm.l.vz.y;
      oi.transform[1*4+3]  = xfm.p.y;
        
      oi.transform[2*4+0]  = xfm.l.vx.z;
      oi.transform[2*4+1]  = xfm.l.vy.z;
      oi.transform[2*4+2]  = xfm.l.vz.z;
      oi.transform[2*4+3]  = xfm.p.z;
        
      oi.flags             = OPTIX_INSTANCE_FLAG_NONE;
      oi.instanceId        = (instanceIDs.empty())?uint32_t(childID):instanceIDs[childID];
      oi.visibilityMask    = (visibilityMasks.empty()) ? 255 : visibilityMasks[childID];
      oi.sbtOffset         = context->numRayTypes * child->getSBTOffset();
      oi.traversableHandle = child->getTraversable(device);
      assert(oi.traversableHandle);
      
      optixInstances[childID] = oi;
    }

    dd.optixInstanceBuffer.alloc(optixInstances.size()*
                                 sizeof(optixInstances[0]));
    dd.optixInstanceBuffer.upload(optixInstances.data(),"optixinstances");
    
    // ==================================================================
    // set up build input
    // ==================================================================
    instanceInput.type
      = OPTIX_BUILD_INPUT_TYPE_INSTANCES;
    instanceInput.instanceArray.instances
      = (hipDeviceptr_t)dd.optixInstanceBuffer.get();
    instanceInput.instanceArray.numInstances
      = (int)optixInstances.size();
      
    // ==================================================================
    // set up accel uptions
    // ==================================================================
    accelOptions.buildFlags = this->buildFlags;

    accelOptions.motionOptions.numKeys = 1;
    if (FULL_REBUILD)
      accelOptions.operation            = OPTIX_BUILD_OPERATION_BUILD;
    else
      accelOptions.operation            = OPTIX_BUILD_OPERATION_UPDATE;
      
    // ==================================================================
    // query build buffer sizes, and allocate those buffers
    // ==================================================================
    OptixAccelBufferSizes blasBufferSizes;
    OPTIX_CHECK(optixAccelComputeMemoryUsage(optixContext,
                                             &accelOptions,
                                             &instanceInput,
                                             1, // num build inputs
                                             &blasBufferSizes
                                             ));
    
    // ==================================================================
    // trigger the build ....
    // ==================================================================
    const size_t tempSize
      = FULL_REBUILD
      ? blasBufferSizes.tempSizeInBytes
      : blasBufferSizes.tempUpdateSizeInBytes;
    LOG("starting to build/refit "
        << prettyNumber(optixInstances.size()) << " instances, "
        << prettyNumber(blasBufferSizes.outputSizeInBytes) << "B in output and "
        << prettyNumber(tempSize) << "B in temp data");
      
    DeviceMemory tempBuffer;
    tempBuffer.alloc(tempSize);
      
    if (FULL_REBUILD) {
      dd.bvhMemory.alloc(blasBufferSizes.outputSizeInBytes);
      dd.memPeak += tempBuffer.size();
      dd.memPeak += dd.bvhMemory.size();
      dd.memFinal = dd.bvhMemory.size();
    }
      
    OPTIX_CHECK(optixAccelBuild(optixContext,
                                /* todo: stream */0,
                                &accelOptions,
                                // array of build inputs:
                                &instanceInput,1,
                                // buffer of temp memory:
                                (hipDeviceptr_t)tempBuffer.get(),
                                tempBuffer.size(),
                                // where we store initial, uncomp bvh:
                                (hipDeviceptr_t)dd.bvhMemory.get(),
                                dd.bvhMemory.size(),
                                /* the traversable we're building: */ 
                                &dd.traversable,
                                /* no compaction for instances: */
                                nullptr,0u
                                ));
      
    OWL_CUDA_SYNC_CHECK();
    
    // ==================================================================
    // aaaaaand .... clean up
    // ==================================================================
    // TODO: move those free's to the destructor, so we can delay the
    // frees until all objects are done
    tempBuffer.free();
      
    LOG_OK("successfully built instance group accel");
  }

  template<bool FULL_REBUILD>
  void InstanceGroup::staticDeviceBuildOn(const DeviceContext::SP &device, LaunchParams::SP launchParams) 
  {
    DeviceData &dd = getDD(device);
    auto optixContext = device->optixContext;

    SetActiveGPU forLifeTime(device);
    LOG("building instance accel over "
        << numChildren << " groups");

    // ==================================================================
    // sanity check that that many instances are actually allowed by optix:
    // ==================================================================
    uint32_t maxInstsPerIAS = 0;
    optixDeviceContextGetProperty
      (optixContext,
       OPTIX_DEVICE_PROPERTY_LIMIT_MAX_INSTANCES_PER_IAS,
       &maxInstsPerIAS,
       sizeof(maxInstsPerIAS));
      
    if (numChildren > maxInstsPerIAS)
      throw std::runtime_error("number of children in instance group exceeds "
                               "OptiX's MAX_INSTANCES_PER_IAS limit");

    if (!FULL_REBUILD && !(buildFlags & OPTIX_BUILD_FLAG_ALLOW_UPDATE))
      throw std::runtime_error("trying to refit an accel struct that was not built with OPTIX_BUILD_FLAG_ALLOW_UPDATE");
    
    if (FULL_REBUILD) {
      dd.memFinal = 0;
      dd.memPeak = 0;
    }
   

    // ==================================================================
    // create instance build inputs on the device
    // ==================================================================

    dd.optixInstanceBuffer.alloc(numChildren*
                                 sizeof(OptixInstance));

    OptixBuildInput              instanceInput  {};
    OptixAccelBuildOptions       accelOptions   {};

    if (!instanceProg.module) {
      OWL_RAISE("instance module is missing - "
                "did you forget to call owlInstanceGroupSetInstanceProg() before"
                " (Instance)GroupAccelBuild()!?");
    }

    // size of each thread block during instance program function call
    vec3i blockDims(32,32,1);
    uint32_t threadsPerBlock = blockDims.x*blockDims.y*blockDims.z;
        
    uint32_t numBlocks = owl::common::divRoundUp((uint32_t)numChildren,threadsPerBlock);
    uint32_t numBlocks_x
      = 1+uint32_t(powf((float)numBlocks,1.f/3.f));
    uint32_t numBlocks_y
      = 1+uint32_t(sqrtf((float)(numBlocks/numBlocks_x)));
    uint32_t numBlocks_z
      = owl::common::divRoundUp(numBlocks,numBlocks_x*numBlocks_y);
        
    vec3i gridDims(numBlocks_x,numBlocks_y,numBlocks_z);

    OptixInstance* d_instances = (OptixInstance*)dd.optixInstanceBuffer.get();

    uint32_t numRayTypes = context->numRayTypes;

    /* arguments for the kernel we are to call */
    void *args[] = {
      &d_instances,
      (void*)&numChildren,
      (void*)&numRayTypes
    };

    hipStream_t stream = device->stream;

    if (launchParams != nullptr) {
      auto &lpDD = launchParams->getDD(device);
      launchParams->writeVariables(lpDD.hostMemory.data(),device);
      
      auto &moduleDD = instanceProg.module->getDD(device);
      // lpDD
      hipDeviceptr_t d_launchDataPtr = 0;
      size_t bytes = 0;      
      _cuModuleGetGlobal(&d_launchDataPtr, &bytes, moduleDD.computeModule, "optixLaunchParams");
      if (d_launchDataPtr == 0) {
        OWL_RAISE("could not find optixLaunchParams in instance program module");
      }
      if (bytes != lpDD.dataSize) {
        OWL_RAISE("size of launch params in instance program module does not match size of launch params in launch params object");
      }
      // now, copy the deviceMemory in the launch params object to this pointer
      // hipError_t rc = hipMemcpyDtoD(d_launchDataPtr, lpDD.deviceMemory.d_pointer, bytes);
#if 1
      hipMemcpy((void*)d_launchDataPtr, (void*)lpDD.hostMemory.ptr, bytes,
                 hipMemcpyHostToDevice);
#else
      hipError_t rc = _cuMemcpyHtoD(d_launchDataPtr, (void*)lpDD.hostMemory.ptr, bytes);

      // Check the result and see if there was an error
      if (rc) {
        const char *errName = 0;
        _cuGetErrorName(rc,&errName);
        OWL_RAISE("CUDA error in copying launch params to instance program module: "
                  +std::string(errName));
      }
#endif
    }

    if (!dd.instanceFuncKernel)
      OWL_RAISE("instance kernel set, but not yet compiled - "
                "did you forget to call BuildPrograms() before"
                " (Instance)GroupAccelBuild()!?");

    hipError_t rc
      = _cuLaunchKernel(dd.instanceFuncKernel,
                       gridDims.x,gridDims.y,gridDims.z,
                       blockDims.x,blockDims.y,blockDims.z,
                       0, stream, args, 0);
    
    if (rc) {
      const char *errName = 0;
      _cuGetErrorName(rc,&errName);
      OWL_RAISE("unknown CUDA error in calling bounds function kernel: "
                +std::string(errName));
    }

    // hipStreamSynchronize(stream);

    // ==================================================================
    // set up build input
    // ==================================================================
    instanceInput.type
      = OPTIX_BUILD_INPUT_TYPE_INSTANCES;
    instanceInput.instanceArray.instances
      = (hipDeviceptr_t)dd.optixInstanceBuffer.get();
    instanceInput.instanceArray.numInstances = (unsigned)numChildren;
      
    // ==================================================================
    // set up accel uptions
    // ==================================================================
    accelOptions.buildFlags = this->buildFlags;

    accelOptions.motionOptions.numKeys = 1;
    if (FULL_REBUILD)
      accelOptions.operation            = OPTIX_BUILD_OPERATION_BUILD;
    else
      accelOptions.operation            = OPTIX_BUILD_OPERATION_UPDATE;
      
    // ==================================================================
    // query build buffer sizes, and allocate those buffers
    // ==================================================================
    OptixAccelBufferSizes blasBufferSizes;
    OPTIX_CHECK(optixAccelComputeMemoryUsage(optixContext,
                                             &accelOptions,
                                             &instanceInput,
                                             1, // num build inputs
                                             &blasBufferSizes
                                             ));
    
    // ==================================================================
    // trigger the build ....
    // ==================================================================
    const size_t tempSize
      = FULL_REBUILD
      ? blasBufferSizes.tempSizeInBytes
      : blasBufferSizes.tempUpdateSizeInBytes;
    LOG("starting to build/refit "
        << prettyNumber(numChildren) << " instances, "
        << prettyNumber(blasBufferSizes.outputSizeInBytes) << "B in output and "
        << prettyNumber(tempSize) << "B in temp data");
      
    DeviceMemory tempBuffer;
    tempBuffer.alloc(tempSize);
      
    if (FULL_REBUILD) {
      dd.bvhMemory.alloc(blasBufferSizes.outputSizeInBytes);
      dd.memPeak += tempBuffer.size();
      dd.memPeak += dd.bvhMemory.size();
      dd.memFinal = dd.bvhMemory.size();
    }
      
    OPTIX_CHECK(optixAccelBuild(optixContext,
                                /* todo: stream */0,
                                &accelOptions,
                                // array of build inputs:
                                &instanceInput,1,
                                // buffer of temp memory:
                                (hipDeviceptr_t)tempBuffer.get(),
                                tempBuffer.size(),
                                // where we store initial, uncomp bvh:
                                (hipDeviceptr_t)dd.bvhMemory.get(),
                                dd.bvhMemory.size(),
                                /* the traversable we're building: */ 
                                &dd.traversable,
                                /* no compaction for instances: */
                                nullptr,0u
                                ));
      
    OWL_CUDA_SYNC_CHECK();
      
    LOG_OK("successfully built instance group accel");
  }

  template<bool FULL_REBUILD>
  void InstanceGroup::motionBlurBuildOn(const DeviceContext::SP &device)
  {
    DeviceData &dd = getDD(device);
    auto optixContext = device->optixContext;
    
    SetActiveGPU forLifeTime(device);
    LOG("building instance accel over "
        << children.size() << " groups");
    
    // ==================================================================
    // sanity check that that many instances are actualy allowed by optix:
    // ==================================================================
    uint32_t maxInstsPerIAS = 0;
    optixDeviceContextGetProperty
      (optixContext,
       OPTIX_DEVICE_PROPERTY_LIMIT_MAX_INSTANCES_PER_IAS,
       &maxInstsPerIAS,
       sizeof(maxInstsPerIAS));
    
    if (children.size() > maxInstsPerIAS)
      throw std::runtime_error("number of children in instnace group exceeds "
                               "OptiX's MAX_INSTANCES_PER_IAS limit");
    
    // ==================================================================
    // build motion transforms
    // ==================================================================
    assert(!transforms[1].empty());
    std::vector<OptixMatrixMotionTransform> motionTransforms(children.size());
#if OPTIX_VERSION >= 70200
    /* since 7.2, optix no longer requires those aabbs (and in fact,
       no longer supports specifying them */
#else
    std::vector<box3f> motionAABBs(children.size());
#endif
    for (size_t childID=0;childID<children.size();childID++) {
      Group::SP child = children[childID];
      assert(child);
      OptixMatrixMotionTransform mt = {};
      mt.child                      = child->getTraversable(device);
      mt.motionOptions.numKeys      = 2;
      mt.motionOptions.timeBegin    = 0.f;
      mt.motionOptions.timeEnd      = 1.f;
      mt.motionOptions.flags        = OPTIX_MOTION_FLAG_NONE;

      for (int timeStep = 0; timeStep < 2; timeStep ++ ) {
        const affine3f xfm = transforms[timeStep][childID];
        mt.transform[timeStep][0*4+0]  = xfm.l.vx.x;
        mt.transform[timeStep][0*4+1]  = xfm.l.vy.x;
        mt.transform[timeStep][0*4+2]  = xfm.l.vz.x;
        mt.transform[timeStep][0*4+3]  = xfm.p.x;
          
        mt.transform[timeStep][1*4+0]  = xfm.l.vx.y;
        mt.transform[timeStep][1*4+1]  = xfm.l.vy.y;
        mt.transform[timeStep][1*4+2]  = xfm.l.vz.y;
        mt.transform[timeStep][1*4+3]  = xfm.p.y;
          
        mt.transform[timeStep][2*4+0]  = xfm.l.vx.z;
        mt.transform[timeStep][2*4+1]  = xfm.l.vy.z;
        mt.transform[timeStep][2*4+2]  = xfm.l.vz.z;
        mt.transform[timeStep][2*4+3]  = xfm.p.z;
      }

      motionTransforms[childID] = mt;

#if OPTIX_VERSION >= 70200
    /* since 7.2, optix no longer requires those aabbs (and in fact,
       no longer supports specifying them */
#else
      motionAABBs[childID]
        = xfmBounds(transforms[0][childID],child->bounds[0]);
      motionAABBs[childID].extend(xfmBounds(transforms[1][childID],child->bounds[1]));
#endif
    }
    // and upload
    dd.motionTransformsBuffer.allocManaged(motionTransforms.size()*
                                    sizeof(motionTransforms[0]));
    dd.motionTransformsBuffer.upload(motionTransforms.data(),"motionTransforms");
      
#if OPTIX_VERSION >= 70200
    /* since 7.2, optix no longer requires those aabbs (and in fact,
       no longer supports specifying them */
#else
    dd.motionAABBsBuffer.allocManaged(motionAABBs.size()*sizeof(box3f));
    dd.motionAABBsBuffer.upload(motionAABBs.data(),"motionaabbs");
#endif      
    // ==================================================================
    // create instance build inputs
    // ==================================================================
    OptixBuildInput              instanceInput  {};
    OptixAccelBuildOptions       accelOptions   {};
      
    //! the N build inputs that go into the builder
    std::vector<OptixInstance>   optixInstances(children.size());

    // now go over all children to set up the buildinputs
    for (size_t childID=0;childID<children.size();childID++) {
      Group::SP child = children[childID];
      assert(child);

      OptixTraversableHandle childMotionHandle = 0;
      OPTIX_CHECK(optixConvertPointerToTraversableHandle
                  (optixContext,
                   (hipDeviceptr_t)(((const uint8_t*)dd.motionTransformsBuffer.get())
                                 +childID*sizeof(motionTransforms[0])
                                 ),
                   OPTIX_TRAVERSABLE_TYPE_MATRIX_MOTION_TRANSFORM,
                   &childMotionHandle));
        
      OptixInstance oi    = {};
      oi.transform[0*4+0]  = 1.f;//xfm.l.vx.x;
      oi.transform[0*4+1]  = 0.f;//xfm.l.vy.x;
      oi.transform[0*4+2]  = 0.f;//xfm.l.vz.x;
      oi.transform[0*4+3]  = 0.f;//xfm.p.x;
        
      oi.transform[1*4+0]  = 0.f;//xfm.l.vx.y;
      oi.transform[1*4+1]  = 1.f;//xfm.l.vy.y;
      oi.transform[1*4+2]  = 0.f;//xfm.l.vz.y;
      oi.transform[1*4+3]  = 0.f;//xfm.p.y;
        
      oi.transform[2*4+0]  = 0.f;//xfm.l.vx.z;
      oi.transform[2*4+1]  = 0.f;//xfm.l.vy.z;
      oi.transform[2*4+2]  = 1.f;//xfm.l.vz.z;
      oi.transform[2*4+3]  = 0.f;//xfm.p.z;
        
      oi.flags             = OPTIX_INSTANCE_FLAG_NONE;
      oi.instanceId        = (instanceIDs.empty())?uint32_t(childID):instanceIDs[childID];
      oi.sbtOffset         = context->numRayTypes * child->getSBTOffset();
      oi.visibilityMask    = (visibilityMasks.empty()) ? 255 : visibilityMasks[childID];
      oi.traversableHandle = childMotionHandle; 
      optixInstances[childID] = oi;
    }

    dd.optixInstanceBuffer.alloc(optixInstances.size()*
                                 sizeof(optixInstances[0]));
    dd.optixInstanceBuffer.upload(optixInstances.data(),"optixinstances");

    // ==================================================================
    // set up build input
    // ==================================================================
    instanceInput.type
      = OPTIX_BUILD_INPUT_TYPE_INSTANCES;
      
    instanceInput.instanceArray.instances
      = dd.optixInstanceBuffer.d_pointer;
    instanceInput.instanceArray.numInstances
      = (int)optixInstances.size();

#if OPTIX_VERSION >= 70200
    /* since 7.2, optix no longer requires those aabbs (and in fact,
       no longer supports specifying them */
#else
    instanceInput.instanceArray.aabbs
      = dd.motionAABBsBuffer.d_pointer;
    instanceInput.instanceArray.numAabbs
      = (int)motionAABBs.size();
#endif
    
      
    // ==================================================================
    // set up accel uption
    // ==================================================================
    accelOptions = {};
    accelOptions.buildFlags =
      OPTIX_BUILD_FLAG_PREFER_FAST_TRACE
      |
      OPTIX_BUILD_FLAG_ALLOW_UPDATE
      ;
    if (FULL_REBUILD)
      accelOptions.operation            = OPTIX_BUILD_OPERATION_BUILD;
    else {
      accelOptions.operation            = OPTIX_BUILD_OPERATION_UPDATE;
    }
      
    // ==================================================================
    // query build buffer sizes, and allocate those buffers
    // ==================================================================
    OptixAccelBufferSizes blasBufferSizes;
    OPTIX_CHECK(optixAccelComputeMemoryUsage(optixContext,
                                             &accelOptions,
                                             &instanceInput,
                                             1, // num build inputs
                                             &blasBufferSizes
                                             ));
    
    // ==================================================================
    // trigger the build ....
    // ==================================================================
    const size_t tempSize
      = FULL_REBUILD
      ? blasBufferSizes.tempSizeInBytes
      : blasBufferSizes.tempUpdateSizeInBytes;
    LOG("starting to build/refit "
        << prettyNumber(optixInstances.size()) << " instances, "
        << prettyNumber(blasBufferSizes.outputSizeInBytes) << "B in output and "
        << prettyNumber(tempSize) << "B in temp data");
      
    DeviceMemory tempBuffer;
    tempBuffer.allocManaged(tempSize);
      
    if (FULL_REBUILD) {
      dd.bvhMemory.alloc(blasBufferSizes.outputSizeInBytes);
    }
      
    OPTIX_CHECK(optixAccelBuild(optixContext,
                                /* todo: stream */0,
                                &accelOptions,
                                // array of build inputs:
                                &instanceInput,1,
                                // buffer of temp memory:
                                (hipDeviceptr_t)tempBuffer.get(),
                                tempBuffer.size(),
                                // where we store initial, uncomp bvh:
                                (hipDeviceptr_t)dd.bvhMemory.get(),
                                dd.bvhMemory.size(),
                                /* the traversable we're building: */ 
                                &dd.traversable,
                                /* no compaction for instances: */
                                nullptr,0u
                                ));

    OWL_CUDA_SYNC_CHECK();
    
    // ==================================================================
    // aaaaaand .... clean up
    // ==================================================================
    // TODO: move those free's to the destructor, so we can delay the
    // frees until all objects are done
    tempBuffer.free();
      
    LOG_OK("successfully built instance group accel");
  }
  
  template<bool FULL_REBUILD>
  void InstanceGroup::motionBlurDeviceBuildOn(const DeviceContext::SP &device, LaunchParams::SP launchParams) 
  {
    OWL_RAISE("Not yet implemented");
  }
} // ::owl
